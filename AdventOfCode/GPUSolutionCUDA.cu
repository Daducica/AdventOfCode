#include "hip/hip_runtime.h"
#include "GPUSolutionCUDA.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

namespace GPUSolution
{
    static __global__ void CheckVisibilityFromLeft_Kernel (bool* visibility, const TreeHeight* forest,
                                                           size_t width, size_t height)
    {
        size_t row {blockIdx.x * blockDim.x + threadIdx.x};
        if (row >= height)
            return;

        TreeHeight maxHeightInRow = -1;
        for (size_t column = 0; column < width; ++column) {
            size_t index = row * width + column;
            const TreeHeight treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromRight_Kernel (bool* visibility, const TreeHeight* forest,
                                                            size_t width, size_t height)
    {
        size_t row {blockIdx.x * blockDim.x + threadIdx.x};
        if (row >= height)
            return;

        TreeHeight maxHeightInRow = -1;
        for (std::int32_t column = std::int32_t (width - 1); column >= 0; --column) {
            size_t index = row * width + column;
            const TreeHeight treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromTop_Kernel (bool* visibility, const TreeHeight* forest,
                                                          size_t width, size_t height)
    {
        size_t column {blockIdx.x * blockDim.x + threadIdx.x};
        if (column >= width)
            return;

        TreeHeight maxHeightInRow = -1;
        for (size_t row = 0; row < height; ++row) {
            size_t index = row * width + column;
            const TreeHeight treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromBottom_Kernel (bool* visibility, const TreeHeight* forest,
                                                             size_t width, size_t height)
    {
        size_t column {blockIdx.x * blockDim.x + threadIdx.x};
        if (column >= width)
            return;

        TreeHeight maxHeightInRow = -1;
        for (std::int32_t row = std::int32_t (height - 1); row >= 0; --row) {
            size_t index = row * width + column;
            const TreeHeight treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static hipError_t CheckVisibilityFromLeft (bool* visibility, const TreeHeight* forest,
                                                size_t width, size_t height)
    {
        size_t arraySize = width * height;

        TreeHeight* deviceForest;
        bool* deviceVisibility;
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice (0);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        cudaStatus = hipMalloc ((void**)&deviceVisibility, arraySize * sizeof (bool));
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc ((void**)&deviceForest, arraySize * sizeof (TreeHeight));
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy (deviceForest, forest, arraySize * sizeof (TreeHeight), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy (deviceVisibility, visibility, arraySize * sizeof (bool), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

        if (height >= 1024 || width >= 1024) {
            printf ("Matrix dimensions are too large.\n");
            goto Error;
        }
        CheckVisibilityFromLeft_Kernel <<<1, (int)height>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromRight_Kernel <<<1, (int)height>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromTop_Kernel <<<1, (int)width>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromBottom_Kernel <<<1, (int)width>>> (deviceVisibility, deviceForest, width, height);

        cudaStatus = hipGetLastError ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "Kernel launch failed: %s\n", hipGetErrorString (cudaStatus));
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy (visibility, deviceVisibility, arraySize * sizeof (bool), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

    Error:
        hipFree (deviceVisibility);
        hipFree (deviceForest);

        return cudaStatus;
    }


    static TreeHeight* Convert2DVectorTo1DArray (const ForestMatrix&forest)
    {
        const size_t height = forest.size ();
        const size_t width = forest[0].size ();
        const size_t arraySize = height * width;

        TreeHeight* array = new TreeHeight[arraySize];
        for (size_t row = 0; row < height; row++) {
            for (size_t col = 0; col < width; col++) {
                array[row*width + col] = forest[row][col];
            }
        }
        return array;
    }


    uint64_t RunVisibleTreeCalculationOnGPU (const ForestMatrix& forest)
    {
        const size_t height = forest.size ();
        const size_t width = forest[0].size ();
        const size_t arraySize = height * width;

        TreeHeight* forestArray = Convert2DVectorTo1DArray (forest);
        bool* visibility = new bool[arraySize];
        std::fill_n (visibility, arraySize, false);

        hipError_t cudaStatus = CheckVisibilityFromLeft (visibility, forestArray, width, height);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "addWithCuda failed!");
            return 1;
        }

        cudaStatus = hipDeviceReset ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipDeviceReset failed!");
            return 1;
        }

        uint64_t visibilityCount = 0;
        for (size_t index = 0; index < arraySize; index++)
            if (visibility[index])
                visibilityCount++;

        delete[] forestArray;
        delete[] visibility;

        return visibilityCount;
    }

}
