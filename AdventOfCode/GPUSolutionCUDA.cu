#include "hip/hip_runtime.h"
#include "GPUSolutionCUDA.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

namespace GPUSolution
{
    static __global__ void CheckVisibilityFromLeft_Kernel (bool* visibility, const short* forest,
                                                           size_t width, size_t height)
    {
        size_t row {blockIdx.x * blockDim.x + threadIdx.x};
        if (row >= height)
            return;

        int maxHeightInRow = -1;
        for (size_t column = 0; column < width; ++column) {
            size_t index = row * width + column;
            const short treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromRight_Kernel (bool* visibility, const short* forest,
                                                            size_t width, size_t height)
    {
        size_t row {blockIdx.x * blockDim.x + threadIdx.x};
        if (row >= height)
            return;

        int maxHeightInRow = -1;
        for (int column = int (width - 1); column >= 0; --column) {
            size_t index = row * width + column;
            const short treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromTop_Kernel (bool* visibility, const short* forest,
                                                          size_t width, size_t height)
    {
        size_t column {blockIdx.x * blockDim.x + threadIdx.x};
        if (column >= width)
            return;

        int maxHeightInRow = -1;
        for (size_t row = 0; row < height; ++row) {
            size_t index = row * width + column;
            const short treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static __global__ void CheckVisibilityFromBottom_Kernel (bool* visibility, const short* forest,
                                                             size_t width, size_t height)
    {
        size_t column {blockIdx.x * blockDim.x + threadIdx.x};
        if (column >= width)
            return;

        int maxHeightInRow = -1;
        for (int row = int (height - 1); row >= 0; --row) {
            size_t index = row * width + column;
            const short treeHeight = forest[index];

            if (treeHeight > maxHeightInRow) {
                maxHeightInRow = treeHeight;
                if (!visibility[index]) {
                    visibility[index] = true;
                }
            }
        }
    }


    static hipError_t CheckVisibilityFromLeft (bool* visibility, const short* forest,
                                                size_t width, size_t height)
    {
        size_t arraySize = width * height;

        short* deviceForest;
        bool* deviceVisibility;
        hipError_t cudaStatus;

        cudaStatus = hipSetDevice (0);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        cudaStatus = hipMalloc ((void**)&deviceVisibility, arraySize * sizeof (bool));
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc ((void**)&deviceForest, arraySize * sizeof (short));
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy (deviceForest, forest, arraySize * sizeof (short), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy (deviceVisibility, visibility, arraySize * sizeof (bool), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

        if (height >= 1024 || width >= 1024) {
            printf ("Matrix dimensions are too large.\n");
            goto Error;
        }
        CheckVisibilityFromLeft_Kernel <<<1, (int)height>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromRight_Kernel <<<1, (int)height>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromTop_Kernel <<<1, (int)width>>> (deviceVisibility, deviceForest, width, height);
        CheckVisibilityFromBottom_Kernel <<<1, (int)width>>> (deviceVisibility, deviceForest, width, height);

        cudaStatus = hipGetLastError ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "Kernel launch failed: %s\n", hipGetErrorString (cudaStatus));
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy (visibility, deviceVisibility, arraySize * sizeof (bool), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipMemcpy failed!");
            goto Error;
        }

    Error:
        hipFree (deviceVisibility);
        hipFree (deviceForest);

        return cudaStatus;
    }


    static short* Convert2DVectorTo1DArray (const std::vector<std::vector<short>>&forest)
    {
        const size_t height = forest.size ();
        const size_t width = forest[0].size ();
        const size_t arraySize = height * width;

        short* array = new short [arraySize];
        for (size_t row = 0; row < height; row++) {
            for (size_t col = 0; col < width; col++) {
                array[row*width + col] = forest[row][col];
            }
        }
        return array;
    }


    uint64_t RunVisibleTreeCalculationOnGPU (const std::vector<std::vector<short>>& forest)
    {
        const size_t height = forest.size ();
        const size_t width = forest[0].size ();
        const size_t arraySize = height * width;

        short* forestArray = Convert2DVectorTo1DArray (forest);
        bool* visibility = new bool[arraySize];
        std::fill_n (visibility, arraySize, false);

        hipError_t cudaStatus = CheckVisibilityFromLeft (visibility, forestArray, width, height);
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "addWithCuda failed!");
            return 1;
        }

        cudaStatus = hipDeviceReset ();
        if (cudaStatus != hipSuccess) {
            fprintf (stderr, "hipDeviceReset failed!");
            return 1;
        }

        uint64_t visibilityCount = 0;
        for (size_t index = 0; index < arraySize; index++)
            if (visibility[index])
                visibilityCount++;

        delete[] forestArray;
        delete[] visibility;

        return visibilityCount;
    }

}
